#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <thrust/device_vector.h>
#include <hipcub/hipcub.hpp>
#include <cuda/std/atomic>

int main()
{
    int deviceCount = cv::cuda::getCudaEnabledDeviceCount();
    std::cout << "CUDA-enabled devices: " << deviceCount << std::endl;

    std::cout << "CUDA info: {}" << cv::getBuildInformation() << std::endl;

    // write a simple cuda matrix compute
    cv::Mat img = cv::imread("fixtures/600speckle.tif");
    cv::cuda::GpuMat dst, src;
    src.upload(img);

    return 0;
}
